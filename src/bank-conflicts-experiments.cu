
#include <hip/hip_runtime.h>
#include <stdio.h>

const int N = 1024000;
const int threadsPerBlock = 1024;
const int blocksPerGrid = (N + threadsPerBlock - 1) / threadsPerBlock;

__global__ void fetch_two_way_conflicts(float *a) {
    __shared__ float BlockCache[threadsPerBlock * 2];
    /* 这样处理相当于将位于全局内存的向量a的N个元素划分成gridDim.x个区块, 每个block内的shared memory缓存自己对应的区块和下一个区块 */
    int tidInBlock = threadIdx.x;
    int baseIdx = blockIdx.x * blockDim.x;
    BlockCache[2*tidInBlock] = a[baseIdx + 2*tidInBlock];
    BlockCache[2*tidInBlock + 1] = a[baseIdx + 2*tidInBlock + 1];
    __syncthreads();  
}

__global__ void fetch_no_conflicts(float *a) {
    __shared__ float BlockCache[threadsPerBlock * 2];
    /* 这样处理相当于将位于全局内存的向量a的N个元素划分成gridDim.x个区块, 每个block内的shared memory缓存自己对应的区块和下一个区块 */
    int tidInBlock = threadIdx.x;
    int baseIdx = blockIdx.x * blockDim.x;
    BlockCache[tidInBlock] = a[baseIdx + tidInBlock];
    BlockCache[tidInBlock + blockDim.x] = a[baseIdx + tidInBlock + blockDim.x];
    __syncthreads();  
}

int main() {
    size_t size = sizeof(float) * N;
    float *host_a;
    host_a = (float*)malloc(size);
    float *dev_a;
    hipMalloc((void**)&dev_a, size);

    for(int i = 0; i < N; i++) {
        host_a[i] = rand() / RAND_MAX;
    }
    hipMemcpy(dev_a, host_a, size, hipMemcpyHostToDevice);

    hipEvent_t start1, stop1, start2, stop2;
    hipEventCreate(&start1);
    hipEventCreate(&stop1);
    hipEventRecord(start1, 0);
    fetch_two_way_conflicts<<<blocksPerGrid, threadsPerBlock>>>(dev_a);
    hipEventRecord(stop1, 0);
    hipEventSynchronize(stop1);
    float time1;
    hipEventElapsedTime(&time1, start1, stop1);
    hipEventDestroy(start1);
    hipEventDestroy(stop1);
    printf("It took %f seconds to fetch data from global memory to shared memory with 2-way bank conflicts\n", time1);

    hipEventCreate(&start2);
    hipEventCreate(&stop2);
    hipEventRecord(start2, 0);
    fetch_no_conflicts<<<blocksPerGrid, threadsPerBlock>>>(dev_a);
    hipEventRecord(stop2, 0);
    hipEventSynchronize(stop2);
    float time2;
    hipEventElapsedTime(&time2, start2, stop2);
    hipEventDestroy(start2);
    hipEventDestroy(stop2);
    printf("It took %f seconds to fetch data from global memory to shared memory with no bank conflicts\n", time2);

    hipFree(dev_a);
    free(host_a);

    return 0;
}